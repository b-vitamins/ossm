#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/extension.h>

namespace ossm {
namespace {

template <typename scalar_t>
__global__ void dlinoss_ex_forward_kernel(const typename scalar_t::value_type* __restrict__ a_diag,
                                          const typename scalar_t::value_type* __restrict__ g_diag,
                                          const typename scalar_t::value_type* __restrict__ step,
                                          const scalar_t* __restrict__ bu_ptr,
                                          scalar_t* __restrict__ out_ptr,
                                          int64_t length,
                                          int64_t batch,
                                          int64_t ssm) {
  using value_t = typename scalar_t::value_type;

  const int64_t series = batch * ssm;
  const int64_t step_stride = series * 2;

  for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < series; idx += blockDim.x * gridDim.x) {
    const int64_t state = idx % ssm;

    const value_t alpha = a_diag[state];
    const value_t gamma = g_diag[state];
    const value_t sigma = step[state];

    const value_t m11 = value_t(1) - sigma * gamma;
    const value_t m12 = -sigma * alpha;
    const value_t m21 = sigma;

    value_t state0_real = value_t(0);
    value_t state0_imag = value_t(0);
    value_t state1_real = value_t(0);
    value_t state1_imag = value_t(0);

    for (int64_t t = 0; t < length; ++t) {
      const int64_t bu_offset = t * series + idx;
      const int64_t out_offset = t * step_stride + idx * 2;

      const scalar_t bu_val = bu_ptr[bu_offset];
      const value_t bu_real = bu_val.real();
      const value_t bu_imag = bu_val.imag();

      const value_t new0_real = state0_real * m11 + state1_real * m12 + bu_real * sigma;
      const value_t new0_imag = state0_imag * m11 + state1_imag * m12 + bu_imag * sigma;
      const value_t new1_real = state0_real * m21 + state1_real;
      const value_t new1_imag = state0_imag * m21 + state1_imag;

      out_ptr[out_offset] = scalar_t(new0_real, new0_imag);
      out_ptr[out_offset + 1] = scalar_t(new1_real, new1_imag);

      state0_real = new0_real;
      state0_imag = new0_imag;
      state1_real = new1_real;
      state1_imag = new1_imag;
    }
  }
}

template <typename scalar_t>
__global__ void dlinoss_ex_backward_kernel(const typename scalar_t::value_type* __restrict__ a_diag,
                                           const typename scalar_t::value_type* __restrict__ g_diag,
                                           const typename scalar_t::value_type* __restrict__ step,
                                           const scalar_t* __restrict__ bu_ptr,
                                           const scalar_t* __restrict__ states_ptr,
                                           const scalar_t* __restrict__ grad_out_ptr,
                                           scalar_t* __restrict__ grad_bu_ptr,
                                           typename scalar_t::value_type* __restrict__ grad_a_ptr,
                                           typename scalar_t::value_type* __restrict__ grad_g_ptr,
                                           typename scalar_t::value_type* __restrict__ grad_step_ptr,
                                           int64_t length,
                                           int64_t batch,
                                           int64_t ssm) {
  using value_t = typename scalar_t::value_type;

  const int64_t series = batch * ssm;
  const int64_t step_stride = series * 2;

  for (int64_t state = blockIdx.x * blockDim.x + threadIdx.x; state < ssm; state += blockDim.x * gridDim.x) {
    const value_t alpha = a_diag[state];
    const value_t gamma = g_diag[state];
    const value_t sigma = step[state];

    const value_t m11 = value_t(1) - sigma * gamma;
    const value_t m12 = -sigma * alpha;
    const value_t m21 = sigma;

    value_t grad_alpha_state = value_t(0);
    value_t grad_gamma_state = value_t(0);
    value_t grad_sigma_state = value_t(0);

    for (int64_t batch_idx = 0; batch_idx < batch; ++batch_idx) {
      const int64_t series_idx = batch_idx * ssm + state;

      value_t grad_state0_real = value_t(0);
      value_t grad_state0_imag = value_t(0);
      value_t grad_state1_real = value_t(0);
      value_t grad_state1_imag = value_t(0);

      const scalar_t* bu_series = bu_ptr + (length - 1) * series + series_idx;
      const scalar_t* grad_out_series = grad_out_ptr + (length - 1) * series + series_idx;
      scalar_t* grad_bu_series = grad_bu_ptr + (length - 1) * series + series_idx;
      const scalar_t* prev_states =
          length > 1 ? states_ptr + (length - 2) * step_stride + series_idx * 2 : nullptr;

      for (int64_t t = length - 1; t >= 0; --t) {
        const scalar_t grad_out_val = *grad_out_series;
        const scalar_t bu_val = *bu_series;

        value_t prev0_real = value_t(0);
        value_t prev0_imag = value_t(0);
        value_t prev1_real = value_t(0);
        value_t prev1_imag = value_t(0);
        if (t > 0 && prev_states != nullptr) {
          const scalar_t prev0_val = prev_states[0];
          const scalar_t prev1_val = prev_states[1];
          prev0_real = prev0_val.real();
          prev0_imag = prev0_val.imag();
          prev1_real = prev1_val.real();
          prev1_imag = prev1_val.imag();
        }

        const value_t grad_out_real = grad_out_val.real();
        const value_t grad_out_imag = grad_out_val.imag();
        const value_t bu_real = bu_val.real();
        const value_t bu_imag = bu_val.imag();

        const value_t grad_new1_real = grad_state1_real + grad_out_real;
        const value_t grad_new1_imag = grad_state1_imag + grad_out_imag;
        const value_t grad_new0_real = grad_state0_real;
        const value_t grad_new0_imag = grad_state0_imag;

        grad_alpha_state += (-sigma) * (grad_new0_real * prev1_real + grad_new0_imag * prev1_imag);
        grad_gamma_state += (-sigma) * (grad_new0_real * prev0_real + grad_new0_imag * prev0_imag);

        const value_t sigma_term_real = prev0_real * (-gamma) + prev1_real * (-alpha) + bu_real;
        const value_t sigma_term_imag = prev0_imag * (-gamma) + prev1_imag * (-alpha) + bu_imag;
        grad_sigma_state += grad_new0_real * sigma_term_real + grad_new0_imag * sigma_term_imag;
        grad_sigma_state += grad_new1_real * prev0_real + grad_new1_imag * prev0_imag;

        const value_t grad_bu_real = grad_new0_real * sigma;
        const value_t grad_bu_imag = grad_new0_imag * sigma;
        *grad_bu_series = scalar_t(grad_bu_real, grad_bu_imag);

        const value_t next_state0_real = grad_new0_real * m11 + grad_new1_real * m21;
        const value_t next_state0_imag = grad_new0_imag * m11 + grad_new1_imag * m21;
        const value_t next_state1_real = grad_new0_real * m12 + grad_new1_real;
        const value_t next_state1_imag = grad_new0_imag * m12 + grad_new1_imag;

        grad_state0_real = next_state0_real;
        grad_state0_imag = next_state0_imag;
        grad_state1_real = next_state1_real;
        grad_state1_imag = next_state1_imag;

        if (t > 0 && prev_states != nullptr) {
          prev_states -= step_stride;
        }
        bu_series -= series;
        grad_out_series -= series;
        grad_bu_series -= series;
      }
    }

    grad_a_ptr[state] = grad_alpha_state;
    grad_g_ptr[state] = grad_gamma_state;
    grad_step_ptr[state] = grad_sigma_state;
  }
}

}  // namespace

void dlinoss_ex_forward_cuda(const at::Tensor& a_diag,
                             const at::Tensor& g_diag,
                             const at::Tensor& step,
                             const at::Tensor& bu,
                             at::Tensor& output) {
  c10::cuda::OptionalCUDAGuard device_guard{bu.device()};

  const auto batch = bu.size(1);
  const auto ssm = bu.size(2);
  const auto series = batch * ssm;

  constexpr int64_t threads = 256;
  const int64_t blocks = std::max<int64_t>(
      1,
      std::min<int64_t>(
          (series + threads - 1) / threads,
          at::cuda::getCurrentDeviceProperties()->maxGridSize[0]));

  AT_DISPATCH_COMPLEX_TYPES(bu.scalar_type(), "dlinoss_ex_forward_cuda", [&] {
    dlinoss_ex_forward_kernel<scalar_t><<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
        a_diag.data_ptr<typename scalar_t::value_type>(),
        g_diag.data_ptr<typename scalar_t::value_type>(),
        step.data_ptr<typename scalar_t::value_type>(),
        bu.data_ptr<scalar_t>(),
        output.data_ptr<scalar_t>(),
        bu.size(0),
        batch,
        ssm);
  });

  AT_CUDA_CHECK(hipGetLastError());
}

void dlinoss_ex_backward_cuda(const at::Tensor& a_diag,
                              const at::Tensor& g_diag,
                              const at::Tensor& step,
                              const at::Tensor& bu,
                              const at::Tensor& states,
                              const at::Tensor& grad_output,
                              at::Tensor& grad_a,
                              at::Tensor& grad_g,
                              at::Tensor& grad_step,
                              at::Tensor& grad_bu) {
  c10::cuda::OptionalCUDAGuard device_guard{bu.device()};

  const auto batch = bu.size(1);
  const auto ssm = bu.size(2);
  const auto series = batch * ssm;

  constexpr int64_t threads = 256;
  const int64_t blocks = std::max<int64_t>(
      1,
      std::min<int64_t>(
          (series + threads - 1) / threads,
          at::cuda::getCurrentDeviceProperties()->maxGridSize[0]));

  AT_DISPATCH_COMPLEX_TYPES(bu.scalar_type(), "dlinoss_ex_backward_cuda", [&] {
    dlinoss_ex_backward_kernel<scalar_t><<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(
        a_diag.data_ptr<typename scalar_t::value_type>(),
        g_diag.data_ptr<typename scalar_t::value_type>(),
        step.data_ptr<typename scalar_t::value_type>(),
        bu.data_ptr<scalar_t>(),
        states.data_ptr<scalar_t>(),
        grad_output.data_ptr<scalar_t>(),
        grad_bu.data_ptr<scalar_t>(),
        grad_a.data_ptr<typename scalar_t::value_type>(),
        grad_g.data_ptr<typename scalar_t::value_type>(),
        grad_step.data_ptr<typename scalar_t::value_type>(),
        bu.size(0),
        batch,
        ssm);
  });

  AT_CUDA_CHECK(hipGetLastError());
}

}  // namespace ossm

